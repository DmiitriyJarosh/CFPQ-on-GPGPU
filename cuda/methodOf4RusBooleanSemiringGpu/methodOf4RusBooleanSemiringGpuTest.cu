#include "hip/hip_runtime.h"

#include "methodOf4RusBooleanSemiringGpu.h"
#include "gpu_memory_management.h"
#include "gpu_timer.cu"
#include<time.h>
#include <assert.h>
#include <stdlib.h>
#include <limits.h>

#define SQUEEZE 32
#define BLOCK_SIZE 32
#define BITS sizeof(__uint32_t) * 8// aka 32

/*
 *squeeze src to dst by 32 in rows
 */
void squeeze_to_bits_rows(const uint32_t *src, int src_rows, int src_cols, uint32_t *dst, int dst_cols) {
    for (int i = 0; i < src_rows; i++) {
        for (int j = 0; j < dst_cols; j++) {
            __uint32_t value = 0;
            for (int n = 0; n < BITS; n++) {
                if (src[i * src_cols + j * BITS + n] != 0) {
                    value |= 1ULL << (31 - n);
                }
            }

            dst[i * dst_cols + j] = value;
        }
    }
}

// dummy mul for testing method of four russian
__global__ void dummy_gpu_semiring_mul(uint32_t *a, uint32_t *b, uint32_t *c, int m, int n, int k) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum |= a[row * n + i] & b[i * k + col];
        }
        c[row * k + col] |= sum;
    }
}

void wrapper_sdummy_semiring_mul(uint32_t *a, uint32_t *b, uint32_t *c, int rows, int cols) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE); 
    dummy_gpu_semiring_mul<<<dimGrid,dimBlock>>>(a, b, c, rows, cols, cols);
    hipDeviceSynchronize();
}
 
void rand_fill(int rows, int sparsity, uint32_t *matrix) {
    for (int i = 0; i < rows * rows; i++) {
        if (rand() % sparsity == 0) {
            matrix[i] = 1;
        }
        else {
            matrix[i] = 0;
        }
    }
}

int method_of_4rus_test(int rows, int table_cols_max, int sparsity) {
    if(rows % SQUEEZE != 0) {
        rows +=  (SQUEEZE - (rows % SQUEEZE));
    }

    int cols = rows / SQUEEZE;
    Tables tables;
    GpuTimer gpuTimer = GpuTimer();
    float elapsedTime;
    tables.initialize(rows,cols, table_cols_max);
    
    uint32_t *unsqueezed_matrixA   = allocate_matrix_host(rows, rows);
    uint32_t *unsqueezed_matrixB   = allocate_matrix_host(rows, rows);
    uint32_t *unsqueezed_matrixC   = allocate_matrix_host(rows, rows);
    uint32_t *unsqueezed_matrixAXB = allocate_matrix_host(rows, rows);

    uint32_t *squeezed_matrixA   = allocate_matrix_host(rows, cols);
    uint32_t *squeezed_matrixB   = allocate_matrix_host(rows, cols);
    uint32_t *squeezed_matrixC   = allocate_matrix_host(rows, cols);    
    uint32_t *squeezed_matrixAXB = allocate_matrix_host(rows, cols);
    
    //rand fill matrices  
    srand(time(NULL));
    rand_fill(rows, sparsity, unsqueezed_matrixA);
    rand_fill(rows, sparsity, unsqueezed_matrixB);
    rand_fill(rows, sparsity, unsqueezed_matrixC);
    for (int i = 0; i < rows * rows; i++) {
        unsqueezed_matrixAXB[i] = unsqueezed_matrixC[i];
    } 

    // device matrices for dummy multiplication
    uint32_t *a_d  = allocate_matrix_device(rows, rows);
    uint32_t *b_d  = allocate_matrix_device(rows, rows);
    uint32_t *axb_d  = allocate_matrix_device(rows, rows);

    copy_host_to_device_sync(unsqueezed_matrixA, a_d, rows * rows);
    copy_host_to_device_sync(unsqueezed_matrixB, b_d, rows * rows);
    copy_host_to_device_sync(unsqueezed_matrixAXB, axb_d, rows * rows);
    
    wrapper_sdummy_semiring_mul(a_d, b_d, axb_d, rows, rows);
    copy_device_to_host_sync(axb_d, unsqueezed_matrixAXB, rows * rows);

    delete_matrix_device(a_d);
    delete_matrix_device(b_d);
    delete_matrix_device(axb_d);
    
    // squeeze dummy matrix as right answer to check correctness of multiplication
    squeeze_to_bits_rows(unsqueezed_matrixAXB, rows, rows, squeezed_matrixAXB, cols);
    
    // squeeze matrices for mul
    squeeze_to_bits_rows(unsqueezed_matrixA, rows, rows, squeezed_matrixA, cols);
    squeeze_to_bits_rows(unsqueezed_matrixC, rows, rows, squeezed_matrixC, cols);   
    squeeze_to_bits_rows(unsqueezed_matrixB, rows, rows, squeezed_matrixB, cols);
    
    
    // allocate device memory for squeezed
    uint32_t * squeezed_matrixB_device = allocate_matrix_device(rows, cols);
    uint32_t * squeezed_matrixA_device = allocate_matrix_device(rows, cols);
    uint32_t * squeezed_matrixC_device = allocate_matrix_device(rows, cols);

    copy_host_to_device_sync(squeezed_matrixB, squeezed_matrixB_device, rows * cols);
    copy_host_to_device_sync(squeezed_matrixA, squeezed_matrixA_device, rows * cols);
    copy_host_to_device_sync(squeezed_matrixC, squeezed_matrixC_device, rows * cols);
     
    gpuTimer.Start();    
    wrapper_method_of_4rus_bool_semiring(squeezed_matrixA_device, squeezed_matrixB_device, 
                                              squeezed_matrixC_device, tables, rows, cols);
    gpuTimer.Stop();
    elapsedTime = gpuTimer.ElapsedMs();
    
    copy_device_to_host_sync(squeezed_matrixC_device, squeezed_matrixC, rows * cols);

    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < rows / SQUEEZE; j++) {
           assert(squeezed_matrixC[i * rows / SQUEEZE + j] == squeezed_matrixAXB[i * rows / SQUEEZE + j]);
       }
    }
    
    delete_matrix_host(unsqueezed_matrixB);
    delete_matrix_host(unsqueezed_matrixA);
    delete_matrix_host(unsqueezed_matrixC);
    delete_matrix_host(unsqueezed_matrixAXB);

    delete_matrix_host(squeezed_matrixAXB);
    delete_matrix_host(squeezed_matrixA);
    delete_matrix_host(squeezed_matrixB);
    delete_matrix_host(squeezed_matrixC);

    delete_matrix_device(squeezed_matrixB_device);
    delete_matrix_device(squeezed_matrixA_device);
    delete_matrix_device(squeezed_matrixC_device);
    tables.free();
    
    printf("Test passed for sparsity=%d, rows = %d is %f ms.\n", sparsity, rows, elapsedTime);
    return 1;
}

/*
* ./program_name initial_size max_size max_sparsity size_step sparsity_step table_size
* 
*
*/
int main(int argc, char *argv[]) {
    int initial_size = strtol(argv[1], NULL, 10);
    int max_size = strtol(argv[2], NULL, 10);
    int max_sparsity = strtol(argv[3], NULL, 10);
    int size_step = strtol(argv[4], NULL, 10);
    int sparsity_step = strtol(argv[5], NULL, 10);
    int table_size = strtol(argv[6], NULL, 10);
    for(int sparsity = 2; sparsity < max_sparsity; sparsity += sparsity_step) {
        for(int size = initial_size; size < max_size; size += size_step) {
            method_of_4rus_test(size, table_size, sparsity); 
        }  
        
    }
}
