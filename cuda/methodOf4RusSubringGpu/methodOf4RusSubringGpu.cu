#include "hip/hip_runtime.h"

#include "methodOf4RusSubringGpu.h"

#define K 8
#define lsb(i) ((i) & -(i)) // return least significant bit
#define BITS sizeof(__uint32_t) * 8// aka 32

__device__ bool is_changed_matrix = false;

//return the next number with the same number of bits
__device__ int snoob(int i) {
    int least = lsb(i);
    int ripple = i + least;
    return (((ripple ^ i) >> 2) / least) | ripple;
}

__global__ void make_table_kernel_subring(uint32_t *B, uint32_t **lookup_tables, int cols,
                                       int rows, int tables_num, int real_cols, int offset) {
    int x_col =  blockIdx.x * BLOCK_SIZE_COL + threadIdx.x;
    int y_row = (blockIdx.y * BLOCK_SIZE_ROW + threadIdx.y) * K;
    int twokey = (1 << K);
    int i;
    int least,rest;

    if(x_col >= cols || y_row >= rows ) {
        //if thread out of current computed part of the table then return
        return;
    }

    //pointer to calculated  table  Br
    uint32_t *T = lookup_tables[blockIdx.y * BLOCK_SIZE_ROW + threadIdx.y];  

    T[x_col] = 0; // row with 000000000...

    // fill when table when 1 bit
    #pragma unroll
    for(int j = 0; j < K; j++) {
        i = 1 << (j);
        T[i * cols + x_col] = B[ (y_row + j) * real_cols  + x_col + offset];
    }
    
    #pragma unroll
    for(int h = 2;h <= K; h++) {
        // fill table elems with h bits in index and <2^K
        // idea taken from Peter Schäfer implementation for m4ri library
        i = (1 << h) - 1;
        for (;i < twokey; i = snoob(i)) {
            least = lsb(i);
            rest = i - least;
            T[i * cols + x_col ] = T[ least * cols + x_col] | T[ rest * cols + x_col];
        }
    }
}

__device__ int get_actual_key(uint32_t composite_key, int j) {
    return  (0xFF) & (composite_key >> (8 * j));
}

__global__ void m4ri_mul_kernel_subring(uint32_t *A, uint32_t *C, uint32_t **lookup_tables,
                                              int rows, int cols, int cols_table, int offset) {
    __shared__ uint32_t local_A[BLOCK_SIZE_ROW][BLOCK_SIZE_COL];
    int col_x = threadIdx.x + blockIdx.x * BLOCK_SIZE_COL + offset;
    int row_y = threadIdx.y + blockIdx.y * BLOCK_SIZE_ROW;
    int col_in_T = threadIdx.x + blockIdx.x * BLOCK_SIZE_COL;
    int full_steps = cols / BLOCK_SIZE_COL;
    int small_step = cols % BLOCK_SIZE_COL;
    uint32_t *T;
    uint32_t composite_key;
    int actual_key;
    uint32_t old_c;

    if(col_x < cols && col_in_T < cols_table && row_y < rows) {
        //if not out 
        old_c = C[row_y * cols + col_x];
    } else {
        old_c = 0;
    }
    
    uint32_t tmp;
    uint32_t value = 0;
    
    #pragma unroll
    for(int i = 0; i < full_steps; i++) {
        // все полные прогоны по ключам
        tmp = __brev(A[ row_y * cols + threadIdx.x + i * BLOCK_SIZE_COL]); // reverse
        local_A[threadIdx.y][threadIdx.x] = tmp;
        __syncthreads();
        
        for(int t = 0; t < BLOCK_SIZE_COL; t++) {
            composite_key = local_A[threadIdx.y][t];
            for(int j = 0; j < 4;j++) {
                T = lookup_tables[BLOCK_SIZE_COL * i * 4 + t * 4 + j];
                actual_key = get_actual_key(composite_key, j);
                value |= T[actual_key * cols_table + col_in_T];
            }
        }
    }
    __syncthreads();
    if(small_step) {
        int cur_step = full_steps;
        if(threadIdx.x + cur_step * BLOCK_SIZE_COL < cols  && row_y < rows){
            tmp = __brev(A[ row_y * cols + threadIdx.x + cur_step * BLOCK_SIZE_COL]); // reverse
            local_A[threadIdx.y][threadIdx.x] = tmp;
        }
        __syncthreads();

        if(col_x >= cols || col_in_T >= cols_table  || row_y >= rows) {
            //threads that out of current part of C contributed to all threads(load keys)
            // and can return
            return;
        }
        
        for(int t = 0; t < small_step; t++) {
            composite_key = local_A[threadIdx.y][t];
            for(int j = 0; j < 4;j++) {
                T = lookup_tables[cur_step * BLOCK_SIZE_COL * 4 + t*4 + j];
                actual_key = get_actual_key(composite_key,j);
                value |= T[actual_key * cols_table + col_in_T];
            }
        }
    }
    value = value | old_c;
    
    if(is_changed_matrix == false && value != old_c) {
        is_changed_matrix = true;
    }

    if(col_x < cols && row_y < rows && col_in_T < cols_table && value != old_c) {
        C[row_y * cols + col_x] = old_c | value;
    }
}

int wrapper_methodOf4Rus_subring(uint32_t *a, uint32_t *b, uint32_t *c, 
                                     Tables tables, int rows, int cols) {
    int is_c_changed = false;
    hipMemcpyToSymbol(HIP_SYMBOL(is_changed_matrix), &is_c_changed, sizeof(bool), 0, hipMemcpyHostToDevice);
    
    //setup configuration for table kernel
    dim3 dimBlock_table_kernel(BLOCK_SIZE_COL,BLOCK_SIZE_ROW);
    
    dim3 dimGrid_table_n   ((tables.cols_n + BLOCK_SIZE_COL-1)/BLOCK_SIZE_COL,
                           (rows + BLOCK_SIZE_ROW * K -1)/(BLOCK_SIZE_ROW * K));
    
    dim3 dimGrid_table_last((tables.cols_last + BLOCK_SIZE_COL-1)/BLOCK_SIZE_COL,
                           (rows + BLOCK_SIZE_ROW * K -1)/(BLOCK_SIZE_ROW * K));
    
    //setup configuration for mul kernel
    dim3 dimBlock_m4ri(BLOCK_SIZE_COL,BLOCK_SIZE_ROW);
    
    dim3 dimGrid_m4ri_n   ( (tables.cols_n + BLOCK_SIZE_COL- 1) / BLOCK_SIZE_COL,
                            ((rows+BLOCK_SIZE_ROW-1)/BLOCK_SIZE_ROW));
    
    dim3 dimGrid_m4ri_last( (tables.cols_last + BLOCK_SIZE_COL- 1) / BLOCK_SIZE_COL,
                            ((rows+BLOCK_SIZE_ROW-1)/BLOCK_SIZE_ROW));
    
    for(int i = 0; i < tables.num_launches; i++) {
        make_table_kernel_subring<<<dimGrid_table_n, dimBlock_table_kernel>>> 
             (b, tables.table_n, tables.cols_n, rows, tables.num_tables, cols, i * tables.cols_n);
        hipDeviceSynchronize();
        m4ri_mul_kernel_subring<<<dimGrid_m4ri_n, dimBlock_m4ri>>>
             (a, c, tables.table_n, rows, cols, tables.cols_n, i * tables.cols_n);
        hipDeviceSynchronize();
    }
    
    if(tables.cols_last != 0) {
        make_table_kernel_subring<<<dimGrid_table_last, dimBlock_table_kernel>>>
             (b, tables.table_last, tables.cols_last, rows, tables.num_tables, cols, 
                                                 tables.num_launches * tables.cols_n);
        hipDeviceSynchronize();
        m4ri_mul_kernel_subring<<<dimGrid_m4ri_last,dimBlock_m4ri>>>
             (a, c, tables.table_last, rows, cols, tables.cols_last, 
                                                   tables.num_launches*tables.cols_n);
        hipDeviceSynchronize();
    }
    
    hipMemcpyFromSymbol(&is_c_changed, HIP_SYMBOL(is_changed_matrix), sizeof(bool), 0, hipMemcpyDeviceToHost);
    return is_c_changed;
}
