
#include "gpu_memory_management.h"
#include <stdio.h>
#include <iostream> 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

uint32_t ** allocate_tables(int num_tables, int num_rows, int num_cols) {    
    uint32_t **d_ppcPtr, *d_pcPtr;
    gpuErrchk(hipMalloc(&d_ppcPtr, sizeof(uint32_t *) * num_tables));

    for(int i = 0; i < num_tables; i ++) {
        gpuErrchk(hipMalloc(&d_pcPtr, sizeof(uint32_t) * num_rows * num_cols));
        gpuErrchk(hipMemset(d_pcPtr, 0, sizeof(uint32_t) * num_rows * num_cols));
        gpuErrchk(hipMemcpy(&d_ppcPtr[i], &d_pcPtr, sizeof(uint32_t *), hipMemcpyHostToDevice));
    }
    return d_ppcPtr;
}

void delete_tables(uint32_t **tables, int num_tables) { 
    uint32_t **someHost;
    gpuErrchk(hipHostMalloc((void **) &someHost, sizeof(uint32_t *) * num_tables)); 
    gpuErrchk(hipMemcpy(someHost, tables, num_tables * sizeof(uint32_t *), hipMemcpyDeviceToHost));

    for(int i = 0; i < num_tables; i ++) {
        gpuErrchk(hipFree((void *) someHost[i]));
    }
    gpuErrchk(hipFree(tables));
    gpuErrchk(hipHostFree(someHost));  
}

uint32_t * allocate_matrix_host(int rows, int cols) {
    uint32_t *matrix;
    gpuErrchk(hipHostMalloc((void **) &matrix, sizeof(uint32_t) * rows * cols));
    return matrix;
}

void delete_matrix_host(uint32_t *matrix) {
    gpuErrchk(hipHostFree(matrix));
}

uint32_t * allocate_matrix_device(int rows, int cols) {
    uint32_t *matrix;
    gpuErrchk(hipMalloc((void **) &matrix, sizeof(uint32_t) * rows * cols));
    return matrix;
}

void delete_matrix_device(uint32_t *matrix) {
    gpuErrchk(hipFree(matrix));
}

void copy_device_to_host_sync(uint32_t *device, uint32_t *host, int elems) {
    gpuErrchk(hipMemcpy(host, device, sizeof(uint32_t) * elems, hipMemcpyDeviceToHost));
}

void copy_host_to_device_sync(uint32_t *host, uint32_t *device, int elems) {
    gpuErrchk(hipMemcpy(device, host, sizeof(uint32_t) * elems, hipMemcpyHostToDevice));
}

void copy_device_to_device_sync(uint32_t *src, uint32_t *dst, int elems) {
    gpuErrchk(hipMemcpy(dst, src, sizeof(uint32_t) * elems, hipMemcpyDeviceToDevice));
}

void copy_device_to_host_async(uint32_t *device, uint32_t *host, int elems) {
    gpuErrchk(hipMemcpyAsync(host, device, sizeof(uint32_t) * elems, hipMemcpyDeviceToHost));
}

void copy_host_to_device_async(uint32_t *host, uint32_t *device, int elems) {
    gpuErrchk(hipMemcpyAsync(device, host, sizeof(uint32_t) * elems, hipMemcpyHostToDevice));
}

void synchronize_with_gpu() {
    hipDeviceSynchronize();
}
